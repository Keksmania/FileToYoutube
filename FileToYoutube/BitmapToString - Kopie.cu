
#include <hip/hip_runtime.h>
extern __global__ void BitmapToString(unsigned char *input, int width, int height, char *output)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int index = y * width + x;

    if (x >= width || y >= height)
        return;

    unsigned char blue = input[index * 3];
    unsigned char green = input[index * 3 + 1];
    output[index] = (char)(blue | (green << 8));
}