
#include <hip/hip_runtime.h>
extern __global__ void BitmapToStringAndFindWhitePixel(unsigned char *input, int width, int height, char *output, int *x_pos, int *y_pos)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int index = y * width + x;

    if (x >= width || y >= height)
        return;

    unsigned char blue = input[index * 3];
    unsigned char green = input[index * 3 + 1];
    unsigned char red = input[index * 3 + 2];
    output[index] = (char)(blue | (green << 8));

    // Check for white pixel
    if (red == 255 && green == 255 && blue == 255)
    {
        *x_pos = x;
        *y_pos = y;
        return;
    }
}